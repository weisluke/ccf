#include "hip/hip_runtime.h"
﻿/******************************************************************************

Please provide credit to Luke Weisenbach should this code be used.
Email: weisluke@alum.mit.edu

******************************************************************************/


#include "complex.cuh"
#include "ccf_microlensing.cuh"
#include "ccf_read_write_files.cuh"
#include "mass_function.cuh"
#include "star.cuh"
#include "util.hpp"

#include <hiprand/hiprand_kernel.h>

#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <limits>
#include <map>
#include <string>


using dtype = double;

/******************************************************************************
constants to be used
******************************************************************************/
const dtype PI = static_cast<dtype>(3.1415926535898);
constexpr int OPTS_SIZE = 2 * 21;
const std::string OPTS[OPTS_SIZE] =
{
	"-h", "--help",
	"-v", "--verbose",
	"-k", "--kappa_tot",
	"-y", "--shear",
	"-s", "--smooth_fraction",
	"-ks", "--kappa_star",
	"-t", "--theta_e",
	"-mf", "--mass_function",
	"-ms", "--m_solar",
	"-ml", "--m_lower",
	"-mh", "--m_upper",
	"-r", "--rectangular",
	"-a", "--approx",
	"-ts", "--taylor",
	"-ns", "--num_stars",
	"-sf", "--starfile",
	"-np", "--num_phi",
	"-nb", "--num_branches",
	"-rs", "--random_seed",
	"-ot", "--outfile_type",
	"-o", "--outfile_prefix"
};
const std::map<std::string, enumMassFunction> MASS_FUNCTIONS{
	{"equal", equal},
	{"uniform", uniform},
	{"salpeter", salpeter},
	{"kroupa", kroupa}
};


/******************************************************************************
default input option values
******************************************************************************/
bool verbose = false;
dtype kappa_tot = static_cast<dtype>(0.3);
dtype shear = static_cast<dtype>(0.3);
dtype smooth_fraction = static_cast<dtype>(0.1);
dtype kappa_star = static_cast<dtype>(0.27);
dtype theta_e = static_cast<dtype>(1);
std::string mass_function_str = "equal";
dtype m_solar = static_cast<dtype>(1);
dtype m_lower = static_cast<dtype>(0.01);
dtype m_upper = static_cast<dtype>(10);
int rectangular = 1;
int approx = 1;
int taylor = 1;
int num_stars = 137;
std::string starfile = "";
int num_phi = 50;
int num_branches = 1;
int random_seed = 0;
std::string outfile_type = ".bin";
std::string outfile_prefix = "./";



/******************************************************************************
Print the program usage help message

\param name -- name of the executable
******************************************************************************/
void display_usage(char* name)
{
	if (name)
	{
		std::cout << "Usage: " << name << " opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	else
	{
		std::cout << "Usage: programname opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	std::cout
		<< "                                                                               \n"
		<< "Options:\n"
		<< "  -h,--help             Show this help message\n"
		<< "  -v,--verbose          Toggle verbose output. Takes no option value.\n"
		<< "  -k,--kappa_tot        Specify the total convergence. Default value: " << kappa_tot << "\n"
		<< "  -y,--shear            Specify the shear. Default value: " << shear << "\n"
		<< "  -s,--smooth_fraction  Specify the fraction of convergence due to smoothly\n"
		<< "                        distributed mass. Default value: " << smooth_fraction << "\n"
		<< "  -ks,--kappa_star      Specify the convergence in point mass lenses. If\n"
		<< "                        provided, this overrides any supplied value for the\n"
		<< "                        smooth fraction. Default value: " << kappa_star << "\n"
		<< "  -t,--theta_e          Specify the size of the Einstein radius of a unit mass\n"
		<< "                        point lens in arbitrary units. Default value: " << theta_e << "\n"
		<< "  -mf,--mass_function   Specify the mass function to use for the point mass\n"
		<< "                        lenses. Options are: equal, uniform, Salpeter, and\n"
		<< "                        Kroupa. Default value: " << mass_function_str << "\n"
		<< "  -ms,--m_solar         Specify the solar mass in arbitrary units.\n"
		<< "                        Default value: " << m_solar << "\n"
		<< "  -ml,--m_lower         Specify the lower mass cutoff in arbitrary units.\n"
		<< "                        Default value: " << m_lower << "\n"
		<< "  -mh,--m_upper         Specify the upper mass cutoff in arbitrary units.\n"
		<< "                        Default value: " << m_upper << "\n"
		<< "  -r,--rectangular      Specify whether the star field should be\n"
		<< "                        rectangular (1) or circular (0). Default value: " << rectangular << "\n"
		<< "  -a,--approx           Specify whether terms for alpha_smooth should be\n"
		<< "                        approximated (1) or exact (0). Default value: " << approx << "\n"
		<< "  -ts,--taylor          Specify the highest degree for the Taylor series of\n"
		<< "                        alpha_smooth. Default value: " << taylor << "\n"
		<< "  -ns,--num_stars       Specify the number of stars desired. Default value: " << num_stars << "\n"
		<< "                        All stars are taken to be of unit mass. If a range of\n"
		<< "                        masses are desired, please input them through a file as\n"
		<< "                        described in the -sf option.\n"
		<< "  -sf,--starfile        Specify the location of a star positions and masses\n"
		<< "                        file. The file may be either a whitespace delimited\n"
		<< "                        text file containing valid values for a star's x\n"
		<< "                        coordinate, y coordinate, and mass, in that order, on\n"
		<< "                        each line, or a binary file of star structures (as\n"
		<< "                        defined in this source code). If specified, the number\n"
		<< "                        of stars is determined through this file and the -ns\n"
		<< "                        option is ignored.\n"
		<< "  -np,--num_phi         Specify the number of steps used to vary phi in the\n"
		<< "                        range [0, 2*pi]. Default value: " << num_phi << "\n"
		<< "  -nb,--num_branches    Specify the number of branches to use for phi in the\n"
		<< "                        range [0, 2*pi]. Default value: " << num_branches << "\n"
		<< "  -rs,--random_seed     Specify the random seed for star field generation. A\n"
		<< "                        value of 0 is reserved for star input files.\n"
		<< "  -ot,--outfile_type    Specify the type of file to be output. Valid options\n"
		<< "                        are binary (.bin) or text (.txt). Default value: " << outfile_type << "\n"
		<< "  -o,--outfile_prefix   Specify the prefix to be used in output filenames.\n"
		<< "                        Default value: " << outfile_prefix << "\n"
		<< "                        Lines of .txt output files are whitespace delimited.\n"
		<< "                        Filenames are:\n"
		<< "                          ccf_parameter_info  various parameter values used in\n"
		<< "                                                calculations\n"
		<< "                          ccf_stars           the first item is num_stars\n"
		<< "                                                followed by binary\n"
		<< "                                                representations of the star\n"
		<< "                                                structures\n"
		<< "                          ccf_ccs             the first item is num_roots and\n"
		<< "                                                the second item is\n"
		<< "                                                num_phi / num_branches + 1\n"
		<< "                                                followed by binary\n"
		<< "                                                representations of the complex\n"
		<< "                                                critical curve values\n"
		<< "                          ccf_caustics        the first item is num_roots and\n"
		<< "                                                the second item is\n"
		<< "                                                num_phi / num_branches + 1\n"
		<< "                                                followed by binary\n"
		<< "                                                representations of the complex\n"
		<< "                                                caustic curve values\n";
}



int main(int argc, char* argv[])
{
	/******************************************************************************
	set precision for printing numbers to screen
	******************************************************************************/
	std::cout.precision(7);

	/******************************************************************************
	if help option has been input, display usage message
	******************************************************************************/
	if (cmd_option_exists(argv, argv + argc, "-h") || cmd_option_exists(argv, argv + argc, "--help"))
	{
		display_usage(argv[0]);
		return -1;
	}

	/******************************************************************************
	if there are input options, but not an even number (since all options take a
	parameter), display usage message and exit
	subtract 1 to take into account that first argument array value is program name
	account for possible verbose option, which is a toggle and takes no input
	******************************************************************************/
	if ((argc - 1) % 2 != 0 &&
		!(cmd_option_exists(argv, argv + argc, "-v") || cmd_option_exists(argv, argv + argc, "--verbose")))
	{
		std::cerr << "Error. Not enough values for options.\n";
		display_usage(argv[0]);
		return -1;
	}

	/******************************************************************************
	check that all options given are valid. use step of 2 since all input options
	take parameters (assumed to be given immediately after the option). start at 1,
	since first array element, argv[0], is program name
	account for possible verbose option, which is a toggle and takes no input
	******************************************************************************/
	for (int i = 1; i < argc; i += 2)
	{
		if (argv[i] == std::string("-v") || argv[i] == std::string("--verbose"))
		{
			verbose = true;
			i--;
			continue;
		}
		if (!cmd_option_valid(OPTS, OPTS + OPTS_SIZE, argv[i]))
		{
			std::cerr << "Error. Invalid input syntax. Unknown option " << argv[i] << "\n";
			display_usage(argv[0]);
			return -1;
		}
	}


	/******************************************************************************
	BEGIN read in options and values, checking correctness and exiting if necessary
	******************************************************************************/

	char* cmdinput = nullptr;

	for (int i = 1; i < argc; i += 2)
	{
		/******************************************************************************
		account for possible verbose option, which is a toggle and takes no input
		******************************************************************************/
		if (argv[i] == std::string("-v") || argv[i] == std::string("--verbose"))
		{
			i--;
			continue;
		}

		cmdinput = cmd_option_value(argv, argv + argc, argv[i]);

		if (argv[i] == std::string("-k") || argv[i] == std::string("--kappa_tot"))
		{
			try
			{
				kappa_tot = static_cast<dtype>(std::stod(cmdinput));
				if (verbose)
				{
					std::cout << "kappa_tot set to: " << kappa_tot << "\n";
				}

			}
			catch (...)
			{
				std::cerr << "Error. Invalid kappa_tot input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-y") || argv[i] == std::string("--shear"))
		{
			try
			{
				shear = static_cast<dtype>(std::stod(cmdinput));
				if (verbose)
				{
					std::cout << "shear set to: " << shear << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid shear input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-s") || argv[i] == std::string("--smooth_fraction"))
		{
			if (cmd_option_exists(argv, argv + argc, "-ks") || cmd_option_exists(argv, argv + argc, "--kappa_star"))
			{
				continue;
			}
			try
			{
				smooth_fraction = static_cast<dtype>(std::stod(cmdinput));
				if (smooth_fraction < 0)
				{
					std::cerr << "Error. Invalid smooth_fraction input. smooth_fraction must be >= 0\n";
					return -1;
				}
				else if (smooth_fraction >= 1)
				{
					std::cerr << "Error. Invalid smooth_fraction input. smooth_fraction must be < 1\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "smooth_fraction set to: " << smooth_fraction << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid smooth_fraction input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ks") || argv[i] == std::string("--kappa_star"))
		{
			try
			{
				kappa_star = static_cast<dtype>(std::stod(cmdinput));
				if (kappa_star < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid kappa_star input. kappa_star must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "kappa_star set to: " << kappa_star << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid kappa_star input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-t") || argv[i] == std::string("--theta_e"))
		{
			try
			{
				theta_e = static_cast<dtype>(std::stod(cmdinput));
				if (theta_e < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid theta_e input. theta_e must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "theta_e set to: " << theta_e << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid theta_e input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-mf") || argv[i] == std::string("--mass_function"))
		{
			mass_function_str = cmdinput;
			make_lowercase(mass_function_str);
			if (!MASS_FUNCTIONS.count(mass_function_str))
			{
				std::cerr << "Error. Invalid mass_function input. mass_function must be equal, uniform, Salpeter, or Kroupa.\n";
				return -1;
			}
			if (verbose)
			{
				std::cout << "mass_function set to: " << mass_function_str << "\n";
			}
		}
		else if (argv[i] == std::string("-ms") || argv[i] == std::string("--m_solar"))
		{
			try
			{
				m_solar = static_cast<dtype>(std::stod(cmdinput));
				if (m_solar < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid m_solar input. m_solar must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "m_solar set to: " << m_solar << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid m_solar input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ml") || argv[i] == std::string("--m_lower"))
		{
			try
			{
				m_lower = static_cast<dtype>(std::stod(cmdinput));
				if (m_lower < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid m_lower input. m_lower must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "m_lower set to: " << m_lower << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid m_lower input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-mh") || argv[i] == std::string("--m_upper"))
		{
			try
			{
				m_upper = static_cast<dtype>(std::stod(cmdinput));
				if (m_upper < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid m_upper input. m_upper must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
				else if (m_upper > std::numeric_limits<dtype>::max())
				{
					std::cerr << "Error. Invalid m_upper input. m_upper must be < " << std::numeric_limits<dtype>::max() << "\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "m_upper set to: " << m_upper << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid m_upper input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-r") || argv[i] == std::string("--rectangular"))
		{
			try
			{
				rectangular = std::stoi(cmdinput);
				if (rectangular != 0 && rectangular != 1)
				{
					std::cerr << "Error. Invalid rectangular input. rectangular must be 1 (rectangular) or 0 (circular).\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "rectangular set to: " << rectangular << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid rectangular input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-a") || argv[i] == std::string("--approx"))
		{
			try
			{
				approx = std::stoi(cmdinput);
				if (approx != 0 && approx != 1)
				{
					std::cerr << "Error. Invalid approx input. approx must be 1 (approximate) or 0 (exact).\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "approx set to: " << approx << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid approx input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ts") || argv[i] == std::string("--taylor"))
		{
			try
			{
				taylor = std::stoi(cmdinput);
				if (taylor < 1)
				{
					std::cerr << "Error. Invalid taylor input. taylor must be an integer > 0\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "taylor set to: " << taylor << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid taylor input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ns") || argv[i] == std::string("--num_stars"))
		{
			try
			{
				num_stars = std::stoi(cmdinput);
				if (num_stars < 1)
				{
					std::cerr << "Error. Invalid num_stars input. num_stars must be an integer > 0\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "num_stars set to: " << num_stars << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_stars input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-sf") || argv[i] == std::string("--starfile"))
		{
			starfile = cmdinput;
			if (verbose)
			{
				std::cout << "starfile set to: " << starfile << "\n";
			}
		}
		else if (argv[i] == std::string("-np") || argv[i] == std::string("--num_phi"))
		{
			try
			{
				num_phi = std::stoi(cmdinput);
				if (num_phi < 1 || num_phi % 2 != 0)
				{
					std::cerr << "Error. Invalid num_phi input. num_phi must be an even integer > 0\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "num_phi set to: " << num_phi << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_phi input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-nb") || argv[i] == std::string("--num_branches"))
		{
			try
			{
				num_branches = std::stoi(cmdinput);
				if (num_branches < 1)
				{
					std::cerr << "Error. Invalid num_branches input. num_branches must be an integer > 0\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "num_branches set to: " << num_branches << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_branches input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-rs") || argv[i] == std::string("--random_seed"))
		{
			try
			{
				random_seed = std::stoi(cmdinput);
				if (random_seed == 0 && !(cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file")))
				{
					std::cerr << "Error. Invalid random_seed input. Seed of 0 is reserved for star input files.\n";
					return -1;
				}
				if (verbose)
				{
					std::cout << "random_seed set to: " << random_seed << "\n";
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid random_seed input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ot") || argv[i] == std::string("--outfile_type"))
		{
			outfile_type = cmdinput;
			make_lowercase(outfile_type);
			if (outfile_type != ".bin" && outfile_type != ".txt")
			{
				std::cerr << "Error. Invalid outfile_type. outfile_type must be .bin or .txt\n";
				return -1;
			}
			if (verbose)
			{
				std::cout << "outfile_type set to: " << outfile_type << "\n";
			}
		}
		else if (argv[i] == std::string("-o") || argv[i] == std::string("--outfile_prefix"))
		{
			outfile_prefix = cmdinput;
			if (verbose)
			{
				std::cout << "outfile_prefix set to: " << outfile_prefix << "\n";
			}
		}
	}
	std::cout << "\n";

	if (num_phi % (2 * num_branches) != 0)
	{
		std::cerr << "Error. Invalid num_phi input. num_phi must be a multiple of 2*num_branches\n";
		return -1;
	}

	if (m_lower >= m_upper)
	{
		std::cerr << "Error. m_lower must be less than m_upper.\n";
		return -1;
	}

	if (cmd_option_exists(argv, argv + argc, "-ks") || cmd_option_exists(argv, argv + argc, "--kappa_star"))
	{
		smooth_fraction = 1 - kappa_star / kappa_tot;
		if (verbose)
		{
			std::cout << "smooth_fraction set to: " << smooth_fraction << "\n\n";
		}
	}
	else
	{
		kappa_star = (1 - smooth_fraction) * kappa_tot;
		if (verbose)
		{
			std::cout << "kappa_star set to: " << kappa_star << "\n\n";
		}
	}

	/******************************************************************************
	END read in options and values, checking correctness and exiting if necessary
	******************************************************************************/


	/******************************************************************************
	check that a CUDA capable device is present
	******************************************************************************/
	int n_devices = 0;

	hipGetDeviceCount(&n_devices);
	if (cuda_error("hipGetDeviceCount", false, __FILE__, __LINE__)) return -1;

	if (verbose)
	{
		std::cout << "Available CUDA capable devices:\n\n";

		for (int i = 0; i < n_devices; i++)
		{
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			if (cuda_error("hipGetDeviceProperties", false, __FILE__, __LINE__)) return -1;

			show_device_info(i, prop);
			std::cout << "\n";
		}
	}

	if (n_devices > 1)
	{
		std::cout << "More than one CUDA capable device detected. Defaulting to first device.\n\n";
	}
	hipSetDevice(0);
	if (cuda_error("hipSetDevice", false, __FILE__, __LINE__)) return -1;


	/******************************************************************************
	determine mass function, <m>, and <m^2>
	******************************************************************************/
	enumMassFunction mass_function = MASS_FUNCTIONS.at(mass_function_str);
	dtype mean_mass = MassFunction<dtype>(mass_function).mean_mass(m_solar, m_lower, m_upper);
	dtype mean_mass2 = MassFunction<dtype>(mass_function).mean_mass2(m_solar, m_lower, m_upper);

	/******************************************************************************
	calculated values for kappa_star, upper and lower mass cutoffs, <m>, and <m^2>
	******************************************************************************/
	dtype kappa_star_actual = static_cast<dtype>(1);
	dtype m_lower_actual = static_cast<dtype>(1);
	dtype m_upper_actual = static_cast<dtype>(1);
	dtype mean_mass_actual = static_cast<dtype>(1);
	dtype mean_mass2_actual = static_cast<dtype>(1);


	/******************************************************************************
	if star file is specified, check validity of values and set num_stars,
	m_lower_actual, m_upper_actual, mean_mass_actual, and mean_mass2_actual based
	on star information
	******************************************************************************/
	if (starfile != "")
	{
		std::cout << "Calculating some parameter values based on star input file " << starfile << "\n";

		if (!read_star_params<dtype>(num_stars, m_lower_actual, m_upper_actual, mean_mass_actual, mean_mass2_actual, starfile))
		{
			std::cerr << "Error. Unable to read star field parameters from file " << starfile << "\n";
			return -1;
		}

		std::cout << "Done calculating some parameter values based on star input file " << starfile << "\n\n";

		m_lower = m_lower_actual;
		m_upper = m_upper_actual;
		mean_mass = mean_mass_actual;
		mean_mass2 = mean_mass2_actual;
	}

	/******************************************************************************
	average magnification of the system
	******************************************************************************/
	dtype mu_ave = 1 / ((1 - kappa_tot) * (1 - kappa_tot) - shear * shear);

	std::cout << "Number of stars used: " << num_stars << "\n\n";

	Complex<dtype> c = std::sqrt(PI * theta_e * theta_e * num_stars * mean_mass / (4 * kappa_star))
		* Complex<dtype>(
			std::sqrt(std::abs((1 - kappa_tot - shear) / (1 - kappa_tot + shear))),
			std::sqrt(std::abs((1 - kappa_tot + shear) / (1 - kappa_tot - shear)))
			);
	dtype rad = std::sqrt(theta_e * theta_e * num_stars * mean_mass / kappa_star);

	/******************************************************************************
	number of roots to be found
	******************************************************************************/
	int num_roots = 2 * num_stars;
	if (rectangular && approx)
	{
		num_roots += static_cast<int>(taylor / 2) * 2;
	}


	/******************************************************************************
	BEGIN memory allocation
	******************************************************************************/

	std::cout << "Beginning memory allocation...\n";

	hiprandState* states = nullptr;
	star<dtype>* stars = nullptr;
	Complex<dtype>* ccs_init = nullptr;
	Complex<dtype>* ccs = nullptr;
	bool* fin = nullptr;
	dtype* errs = nullptr;
	int* has_nan = nullptr;
	Complex<dtype>* caustics = nullptr;

	/******************************************************************************
	allocate memory for stars
	******************************************************************************/
	hipMallocManaged(&states, num_stars * sizeof(hiprandState));
	if (cuda_error("hipMallocManaged(*states)", false, __FILE__, __LINE__)) return -1;
	hipMallocManaged(&stars, num_stars * sizeof(star<dtype>));
	if (cuda_error("hipMallocManaged(*stars)", false, __FILE__, __LINE__)) return -1;

	/******************************************************************************
	allocate memory for array of critical curve positions
	******************************************************************************/
	hipMallocManaged(&ccs_init, (num_phi + num_branches) * num_roots * sizeof(Complex<dtype>));
	if (cuda_error("hipMallocManaged(*ccs_init)", false, __FILE__, __LINE__)) return -1;

	/******************************************************************************
	allocate memory for array of transposed critical curve positions
	******************************************************************************/
	hipMallocManaged(&ccs, (num_phi + num_branches) * num_roots * sizeof(Complex<dtype>));
	if (cuda_error("hipMallocManaged(*ccs)", false, __FILE__, __LINE__)) return -1;

	/******************************************************************************
	array to hold t/f values of whether or not roots have been found to desired
	precision
	******************************************************************************/
	hipMallocManaged(&fin, num_branches * 2 * num_roots * sizeof(bool));
	if (cuda_error("hipMallocManaged(*fin)", false, __FILE__, __LINE__)) return -1;

	/******************************************************************************
	array to hold root errors
	******************************************************************************/
	hipMallocManaged(&errs, (num_phi + num_branches) * num_roots * sizeof(dtype));
	if (cuda_error("hipMallocManaged(*errs)", false, __FILE__, __LINE__)) return -1;

	/******************************************************************************
	variable to hold whether array of root errors has nan errors or not
	******************************************************************************/
	hipMallocManaged(&has_nan, sizeof(int));
	if (cuda_error("hipMallocManaged(*has_nan)", false, __FILE__, __LINE__)) return -1;

	/******************************************************************************
	array to hold caustic positions
	******************************************************************************/
	hipMallocManaged(&caustics, (num_phi + num_branches) * num_roots * sizeof(Complex<dtype>));
	if (cuda_error("hipMallocManaged(*caustics)", false, __FILE__, __LINE__)) return -1;

	std::cout << "Done allocating memory.\n\n";

	/******************************************************************************
	END memory allocation
	******************************************************************************/


	/******************************************************************************
	variables for kernel threads and blocks
	******************************************************************************/
	dim3 threads;
	dim3 blocks;

	/******************************************************************************
	number of threads per block, and number of blocks per grid
	uses 512 for number of threads in x dimension, as 1024 is the maximum allowable
	number of threads per block but is too large for some memory allocation, and
	512 is next power of 2 smaller
	******************************************************************************/
	set_threads(threads, 512);
	set_blocks(threads, blocks, num_stars);


	/******************************************************************************
	BEGIN populating star array
	******************************************************************************/

	if (starfile == "")
	{
		std::cout << "Generating star field...\n";

		/******************************************************************************
		if random seed was not provided, get one based on the time
		******************************************************************************/
		if (random_seed == 0)
		{
			random_seed = static_cast<int>(std::chrono::system_clock::now().time_since_epoch().count());
		}

		/******************************************************************************
		generate random star field if no star file has been given
		******************************************************************************/
		initialize_curand_states_kernel<dtype> <<<blocks, threads>>> (states, num_stars, random_seed);
		if (cuda_error("initialize_curand_states_kernel", true, __FILE__, __LINE__)) return -1;
		if (rectangular)
		{
			generate_rectangular_star_field_kernel<dtype> <<<blocks, threads>>> (states, stars, num_stars, c, mass_function, m_solar, m_lower, m_upper);
		}
		else
		{
			generate_circular_star_field_kernel<dtype> <<<blocks, threads>>> (states, stars, num_stars, rad, mass_function, m_solar, m_lower, m_upper);
		}
		if (cuda_error("generate_star_field_kernel", true, __FILE__, __LINE__)) return -1;

		std::cout << "Done generating star field.\n\n";

		/******************************************************************************
		calculate kappa_star_actual, m_lower_actual, m_upper_actual, mean_mass_actual,
		and mean_mass2_actual based on star information
		******************************************************************************/
		calculate_star_params<dtype>(stars, num_stars, m_lower_actual, m_upper_actual, mean_mass_actual, mean_mass2_actual);
		if (rectangular)
		{
			kappa_star_actual = PI * theta_e * theta_e * num_stars * mean_mass_actual / (4 * c.re * c.im);
		}
		else
		{
			kappa_star_actual = theta_e * theta_e * num_stars * mean_mass_actual / (rad * rad);
		}
	}
	else
	{
		/******************************************************************************
		ensure random seed is 0 to denote that stars come from external file
		******************************************************************************/
		random_seed = 0;

		std::cout << "Reading star field from file " << starfile << "\n";

		/******************************************************************************
		reading star field from external file
		******************************************************************************/
		if (!read_star_file<dtype>(stars, num_stars, starfile))
		{
			std::cerr << "Error. Unable to read star field from file " << starfile << "\n";
			return -1;
		}

		std::cout << "Done reading star field from file " << starfile << "\n\n";
	}

	/************************
	END populating star array
	************************/


	/******************************************************************************
	redefine thread and block size to maximize parallelization
	******************************************************************************/
	set_threads(threads, 32);
	set_blocks(threads, blocks, num_roots, 2, num_branches);


	/******************************************************************************
	set boolean (int) of errors having nan values to false (0)
	******************************************************************************/
	*has_nan = 0;

	/******************************************************************************
	initialize roots for centers of all branches to lie at starpos +/- 1
	******************************************************************************/
	if (verbose)
	{
		std::cout << "Initializing root positions...\n";
	}
	for (int j = 0; j < num_branches; j++)
	{
		int center = (num_phi / (2 * num_branches) + j * num_phi / num_branches + j) * num_roots;
		for (int i = 0; i < num_stars; i++)
		{
			ccs_init[center + i] = stars[i].position + 1;
			ccs_init[center + i + num_stars] = stars[i].position - 1;
		}
		if (rectangular && approx)
		{
			int nroots_extra = static_cast<int>(taylor / 2) * 2;
			for (int i = 0; i < nroots_extra; i++)
			{
				ccs_init[center + 2 * num_stars + i] = c.abs() * 
					Complex<dtype>(std::cos(2 * PI / nroots_extra * i), std::sin(2 * PI / nroots_extra * i));
			}
		}
	}
	if (verbose)
	{
		std::cout << "Done initializing root positions.\n\n";
	}

	/******************************************************************************
	initialize values of whether roots have been found to false
	twice the number of roots for a single value of phi for each branch, times the
	number of branches, because we will be growing roots for two values of phi
	simultaneously for each branch
	******************************************************************************/
	for (int i = 0; i < num_branches * 2 * num_roots; i++)
	{
		fin[i] = false;
	}

	for (int i = 0; i < (num_phi + num_branches) * num_roots; i++)
	{
		errs[i] = static_cast<dtype>(0);
	}

	/******************************************************************************
	number of iterations to use for root finding
	empirically, 30 seems to be roughly the amount needed
	******************************************************************************/
	int num_iters = 30;


	/******************************************************************************
	start and end time for timing purposes
	******************************************************************************/
	std::chrono::high_resolution_clock::time_point starttime;
	std::chrono::high_resolution_clock::time_point endtime;


	/******************************************************************************
	begin finding initial roots and calculate time taken in seconds
	******************************************************************************/
	std::cout << "Finding initial roots...\n";
	starttime = std::chrono::high_resolution_clock::now();

	/******************************************************************************
	each iteration of this loop calculates updated positions of all roots for the
	center of each branch in parallel
	ideally, the number of loop iterations is enough to ensure that all roots are
	found to the desired accuracy
	******************************************************************************/
	for (int i = 0; i < num_iters; i++)
	{
		/******************************************************************************
		display percentage done
		******************************************************************************/
		print_progress(i, num_iters - 1);

		find_critical_curve_roots_kernel<dtype> <<<blocks, threads>>> (kappa_tot, shear, theta_e, stars, num_stars, kappa_star, 
			rectangular, c, approx, taylor, ccs_init, num_roots, 0, num_phi, num_branches, fin);
		if (cuda_error("find_critical_curve_roots_kernel", true, __FILE__, __LINE__)) return -1;
	}
	endtime = std::chrono::high_resolution_clock::now();
	double t_init_roots = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;

	std::cout << "\nDone finding roots. Elapsed time: " << t_init_roots << " seconds.\n";


	/******************************************************************************
	calculate errors in 1/mu for initial roots
	******************************************************************************/
	if (verbose)
	{
		std::cout << "Calculating maximum errors in 1/mu...\n";
	}
	find_errors_kernel<dtype> <<<blocks, threads>>> (ccs_init, num_roots, kappa_tot, shear, theta_e, stars, num_stars, kappa_star, 
		rectangular, c, approx, taylor, 0, num_phi, num_branches, errs);
	if (cuda_error("find_errors_kernel", false, __FILE__, __LINE__)) return -1;

	has_nan_err_kernel<dtype> <<<blocks, threads>>> (errs, (num_phi + num_branches) * num_roots, has_nan);
	if (cuda_error("has_nan_err_kernel", true, __FILE__, __LINE__)) return -1;

	if (*has_nan)
	{
		std::cerr << "Error. Errors in 1/mu contain values which are not positive real numbers.\n";
		return -1;
	}

	/******************************************************************************
	find max error and print
	must be performed in loops as CUDA does not currently have an atomicMax for
	floats or doubles, only ints
	******************************************************************************/
	int num_errs = (num_phi + num_branches) * num_roots;
	while (num_errs > 1)
	{
		if (num_errs % 2 != 0)
		{
			errs[num_errs - 2] = std::fmax(errs[num_errs - 2], errs[num_errs - 1]);
			num_errs -= 1;
		}
		num_errs /= 2;
		max_err_kernel<dtype> <<<blocks, threads>>> (errs, num_errs);
		if (cuda_error("max_err_kernel", true, __FILE__, __LINE__)) return -1;
	}
	dtype max_error = errs[0];
	if (verbose)
	{
		std::cout << "Done calculating maximum errors in 1/mu.\n";
	}
	std::cout << "Maximum error in 1/mu: " << max_error << "\n\n";


	/******************************************************************************
	reduce number of iterations needed, as roots should stay close to previous
	positions
	******************************************************************************/
	num_iters = 20;


	/******************************************************************************
	begin finding critical curves and calculate time taken in seconds
	******************************************************************************/
	std::cout << "Finding critical curve positions...\n";
	starttime = std::chrono::high_resolution_clock::now();

	/******************************************************************************
	the outer loop will step through different values of phi
	we use num_phi/(2*num_branches) steps, as we will be working our way out from
	the middle of each branch for the array of roots simultaneously
	******************************************************************************/
	for (int j = 1; j <= num_phi / (2 * num_branches); j++)
	{
		/******************************************************************************
		set critical curve array elements to be equal to last roots
		fin array is reused each time
		******************************************************************************/
		prepare_roots_kernel<dtype> <<<blocks, threads>>> (ccs_init, num_roots, j, num_phi, num_branches, fin);
		if (cuda_error("prepare_roots_kernel", false, __FILE__, __LINE__)) return -1;

		/******************************************************************************
		calculate roots for current values of j
		******************************************************************************/
		for (int i = 0; i < num_iters; i++)
		{
			find_critical_curve_roots_kernel<dtype> <<<blocks, threads>>> (kappa_tot, shear, theta_e, stars, num_stars, kappa_star, 
				rectangular, c, approx, taylor, ccs_init, num_roots, j, num_phi, num_branches, fin);
			if (cuda_error("find_critical_curve_roots_kernel", false, __FILE__, __LINE__)) return -1;
		}
		/******************************************************************************
		only perform synchronization call after roots have all been found
		this allows the print_progress call in the outer loop to accurately display the
		amount of work done so far
		one could move the synchronization call outside of the outer loop for a slight
		speed-up, at the cost of not knowing how far along in the process the
		computations have gone
		******************************************************************************/
		if (j * 100 / (num_phi / (2 * num_branches)) > (j - 1) * 100 / (num_phi / (2 * num_branches)))
		{
			hipDeviceSynchronize();
			if (cuda_error("hipDeviceSynchronize", false, __FILE__, __LINE__)) return -1;
			print_progress(j, num_phi / (2 * num_branches));
		}
	}

	endtime = std::chrono::high_resolution_clock::now();
	double t_ccs = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
	std::cout << "\nDone finding critical curve positions. Elapsed time: " << t_ccs << " seconds.\n\n";


	/******************************************************************************
	find max error in 1/mu over whole critical curve array and print
	******************************************************************************/
	std::cout << "Finding maximum error in 1/mu over all calculated critical curve positions...\n";

	for (int j = 0; j <= num_phi / (2 * num_branches); j++)
	{
		find_errors_kernel<dtype> <<<blocks, threads>>> (ccs_init, num_roots, kappa_tot, shear, theta_e, stars, num_stars, kappa_star, 
			rectangular, c, approx, taylor, j, num_phi, num_branches, errs);
		if (cuda_error("find_errors_kernel", false, __FILE__, __LINE__)) return -1;
	}

	has_nan_err_kernel<dtype> <<<blocks, threads>>> (errs, (num_phi + num_branches) * num_roots, has_nan);
	if (cuda_error("has_nan_err_kernel", true, __FILE__, __LINE__)) return -1;

	if (*has_nan)
	{
		std::cerr << "Error. Errors in 1/mu contain values which are not positive real numbers.\n";
		return -1;
	}

	num_errs = (num_phi + num_branches) * num_roots;
	while (num_errs > 1)
	{
		if (num_errs % 2 != 0)
		{
			errs[num_errs - 2] = std::fmax(errs[num_errs - 2], errs[num_errs - 1]);
			num_errs -= 1;
		}
		num_errs /= 2;
		max_err_kernel<dtype> <<<blocks, threads>>> (errs, num_errs);
		if (cuda_error("max_err_kernel", true, __FILE__, __LINE__)) return -1;
	}
	max_error = errs[0];
	std::cout << "Maximum error in 1/mu: " << max_error << "\n\n";


	/******************************************************************************
	redefine thread and block size to maximize parallelization
	******************************************************************************/
	set_threads(threads, 512);
	set_blocks(threads, blocks, num_roots * (num_phi + num_branches));

	if (verbose)
	{
		std::cout << "Transposing critical curve array...\n";
	}
	starttime = std::chrono::high_resolution_clock::now();
	transpose_array_kernel<dtype> <<<blocks, threads>>> (ccs_init, (num_phi + num_branches), num_roots, ccs);
	if (cuda_error("transpose_array_kernel", true, __FILE__, __LINE__)) return -1;
	endtime = std::chrono::high_resolution_clock::now();
	if (verbose)
	{
		std::cout << "Done transposing critical curve array. Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0 << " seconds.\n\n";
	}
	
	std::cout << "Finding caustic positions...\n";
	starttime = std::chrono::high_resolution_clock::now();
	find_caustics_kernel<dtype> <<<blocks, threads>>> (ccs, (num_phi + num_branches) * num_roots, kappa_tot, shear, theta_e, stars, num_stars, kappa_star, 
		rectangular, c, approx, taylor, caustics);
	if (cuda_error("find_caustics_kernel", true, __FILE__, __LINE__)) return -1;
	endtime = std::chrono::high_resolution_clock::now();
	double t_caustics = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
	std::cout << "Done finding caustic positions. Elapsed time: " << t_caustics << " seconds.\n\n";



	/******************************************************************************
	stream for writing output files
	set precision to 9 digits
	******************************************************************************/
	std::ofstream outfile;
	outfile.precision(9);
	std::string fname;


	std::cout << "Writing parameter info...\n";
	fname = outfile_prefix + "ccf_parameter_info.txt";
	outfile.open(fname);
	if (!outfile.is_open())
	{
		std::cerr << "Error. Failed to open file " << fname << "\n";
		return -1;
	}
	outfile << "kappa_tot " << kappa_tot << "\n";
	outfile << "shear " << shear << "\n";
	outfile << "mu_ave " << mu_ave << "\n";
	outfile << "smooth_fraction " << smooth_fraction << "\n";
	outfile << "kappa_star " << kappa_star << "\n";
	if (starfile == "")
	{
		outfile << "kappa_star_actual " << kappa_star_actual << "\n";
	}
	outfile << "theta_e " << theta_e << "\n";
	if (starfile == "")
	{
		outfile << "mass_function " << mass_function_str << "\n";
		if (mass_function_str == "salpeter" || mass_function_str == "kroupa")
		{
			outfile << "m_solar " << m_solar << "\n";
		}
		if (mass_function_str != "equal")
		{
			outfile << "m_lower " << m_lower << "\n";
			outfile << "m_upper " << m_upper << "\n";
			outfile << "m_lower_actual " << m_lower_actual << "\n";
			outfile << "m_upper_actual " << m_upper_actual << "\n";
		}
		outfile << "mean_mass " << mean_mass << "\n";
		outfile << "mean_mass2 " << mean_mass2 << "\n";
		if (mass_function_str != "equal")
		{
			outfile << "mean_mass_actual " << mean_mass_actual << "\n";
			outfile << "mean_mass2_actual " << mean_mass2_actual << "\n";
		}
	}
	else
	{
		outfile << "m_lower_actual " << m_lower_actual << "\n";
		outfile << "m_upper_actual " << m_upper_actual << "\n";
		outfile << "mean_mass_actual " << mean_mass_actual << "\n";
		outfile << "mean_mass2_actual " << mean_mass2_actual << "\n";
	}
	outfile << "num_stars " << num_stars << "\n";
	if (rectangular)
	{
		outfile << "corner_x1 " << c.re << "\n";
		outfile << "corner_x2 " << c.im << "\n";
		if (approx)
		{
			outfile << "taylor " << taylor << "\n";
		}
	}
	else
	{
		outfile << "rad " << rad << "\n";
	}
	outfile << "num_roots " << num_roots << "\n";
	outfile << "num_phi " << num_phi << "\n";
	outfile << "num_branches " << num_branches << "\n";
	outfile << "random_seed " << random_seed << "\n";
	outfile << "max_error_1/mu " << max_error << "\n";
	outfile << "t_init_roots " << t_init_roots << "\n";
	outfile << "t_ccs " << t_ccs << "\n";
	outfile << "t_caustics " << t_caustics << "\n";
	outfile.close();
	std::cout << "Done writing parameter info to file " << fname << "\n\n";


	std::cout << "Writing star info...\n";
	fname = outfile_prefix + "ccf_stars" + outfile_type;
	if (!write_star_file<dtype>(stars, num_stars, fname))
	{
		std::cerr << "Error. Unable to write star info to file " << fname << "\n";
		return -1;
	}
	std::cout << "Done writing star info to file " << fname << "\n\n";


	/******************************************************************************
	write critical curve positions
	******************************************************************************/
	std::cout << "Writing critical curve positions...\n";
	if (outfile_type == ".txt")
	{
		fname = outfile_prefix + "ccf_ccs_x" + outfile_type;
		if (!write_re_array<dtype>(ccs, num_roots * num_branches, num_phi / num_branches + 1, fname))
		{
			std::cerr << "Error. Unable to write ccs x info to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing critical curve x positions to file " << fname << "\n";

		fname = outfile_prefix + "ccf_ccs_y" + outfile_type;
		if (!write_im_array<dtype>(ccs, num_roots * num_branches, num_phi / num_branches + 1, fname))
		{
			std::cerr << "Error. Unable to write ccs y info to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing critical curve y positions to file " << fname << "\n";
	}
	else
	{
		fname = outfile_prefix + "ccf_ccs" + outfile_type;
		if (!write_complex_array<dtype>(ccs, num_roots * num_branches, num_phi / num_branches + 1, fname))
		{
			std::cerr << "Error. Unable to write ccs info to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing critical curve positions to file " << fname << "\n";
	}
	std::cout << "\n";


	/******************************************************************************
	write caustic positions
	******************************************************************************/
	std::cout << "Writing caustic positions...\n";
	if (outfile_type == ".txt")
	{
		fname = outfile_prefix + "ccf_caustics_x" + outfile_type;
		if (!write_re_array<dtype>(caustics, num_roots * num_branches, num_phi / num_branches + 1, fname))
		{
			std::cerr << "Error. Unable to write caustic x info to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing caustic x positions to file " << fname << "\n";

		fname = outfile_prefix + "ccf_caustics_y" + outfile_type;
		if (!write_im_array<dtype>(caustics, num_roots * num_branches, num_phi / num_branches + 1, fname))
		{
			std::cerr << "Error. Unable to write caustic y info to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing caustic y positions to file " << fname << "\n";
	}
	else
	{
		fname = outfile_prefix + "ccf_caustics" + outfile_type;
		if (!write_complex_array<dtype>(caustics, num_roots * num_branches, num_phi / num_branches + 1, fname))
		{
			std::cerr << "Error. Unable to write caustic info to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing caustic positions to file " << fname << "\n";
	}
	std::cout << "\n";

	std::cout << "Done.\n";

	hipDeviceReset();
	if (cuda_error("hipDeviceReset", false, __FILE__, __LINE__)) return -1;

	return 0;
}

